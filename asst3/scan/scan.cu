#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"
// #define DEBUG
#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result

__global__ void scan_up(int* result, int d) {
    int idx_plus_1 = (blockIdx.x * blockDim.x + threadIdx.x + 1) * d;
    result[idx_plus_1 - 1] += result[idx_plus_1 - d / 2 - 1];
}

__global__ void reset_zero(int* result, int N) {
    result[N - 1] = 0;
}

__global__ void scan_down(int* result, int d) {
    int idx_plus_1 = (blockIdx.x * blockDim.x + threadIdx.x + 1) * d;
        int t = result[idx_plus_1 - 1];
        result[idx_plus_1 - 1] += result[idx_plus_1 - d / 2 - 1];
        result[idx_plus_1 - d / 2 - 1] = t;
}

std::pair<int, int> get_param(int N, int d) {
        int thread_num = N / d;
        int num_blocks;
        int thread_per_block;
        // 如果需要的线程数少于THREADS_PER_BLOCK，就分配一个block
        if (thread_num <= THREADS_PER_BLOCK) {
            num_blocks = 1;
            thread_per_block = thread_num;
        } else {
            num_blocks = (thread_num + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
            thread_per_block = THREADS_PER_BLOCK;
        }
        return {num_blocks, thread_per_block};
}


void exclusive_scan(int* input, int N, int* result)
{
    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep input
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    N = nextPow2(N);
    // int num_blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    // up
    for (int d = 2; d <= N; d *= 2) {
        auto param = get_param(N, d);
        #ifdef DEBUG
        printf("%d blocks, %d thread !\n", param.first, param.second);
        #endif
        scan_up<<<param.first, param.second>>>(result, d);
        hipDeviceSynchronize();
    }

    reset_zero<<<1, 1>>>(result, N);

    hipDeviceSynchronize();

    for (int d = N; d >= 2; d /= 2) {
        auto param = get_param(N, d);
        #ifdef DEBUG
        printf("%d blocks, %d thread !\n", param.first, param.second);
        #endif
        scan_down<<<param.first, param.second>>>(result, d);
        hipDeviceSynchronize();
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration;
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration;
}


__global__ void comp(int* input, int* output, int N) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N - 1) {
        output[idx] = input[idx] == input[idx + 1];
    } else {
        output[idx] = 0;
    }
    #ifdef DEBUG
    if (output[idx] == 1) {
        printf("cmp: idx: %d, value: %d\n", idx, input[idx]);
    }
    #endif
}

__global__ void fill(int* input, int* prefix_idx, int N, int *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N - 1 && prefix_idx[idx + 1] > prefix_idx[idx]) {
        output[prefix_idx[idx]] = idx;
        #ifdef DEBUG
        printf("fill: idx: %d, prefix_idx: %d, value: %d\n", idx, prefix_idx[idx], input[idx]);
        #endif
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int len_pow2 = nextPow2(length);

    int* prefix_idx;

    int num_block;
    int threads_per_block;
    if (len_pow2 <= THREADS_PER_BLOCK) {
        num_block = 1;
        threads_per_block = len_pow2;
    }  else {
        num_block = (len_pow2 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        threads_per_block = THREADS_PER_BLOCK;
    }

    #ifdef DEBUG
    printf("block: %d, thread: %d\n", num_block, threads_per_block);
    #endif

    hipMalloc(&prefix_idx, len_pow2 * sizeof(int));
    comp<<<num_block, threads_per_block>>>(device_input, prefix_idx, length);  // 求每一位是否满足条件
    hipDeviceSynchronize();
    exclusive_scan(prefix_idx, len_pow2 ,prefix_idx); // 求前缀和，代表在结果中的下标
    fill<<<num_block, threads_per_block>>>(device_input, prefix_idx, len_pow2, device_output);
    hipDeviceSynchronize();
    int ret;
    hipMemcpy(&ret, prefix_idx + length - 1, sizeof(int), hipMemcpyDeviceToHost);
    return ret;
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();

    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime;
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
